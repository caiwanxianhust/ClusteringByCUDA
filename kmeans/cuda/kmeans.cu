#include "hip/hip_runtime.h"
#include "kmeans.h"
#include "cuda_kernels.cuh"

#include <stdio.h>

namespace clustering {

template <typename DataType>
KmeansGPU<DataType>::KmeansGPU(DataType *h_clusters, int num_clusters, int num_features, int num_samples, int max_iters) 
    : m_num_clusters(num_clusters), m_num_features(num_features), m_num_samples(num_samples), m_max_iters(max_iters), 
    m_optTarget(1e7f), m_eplison(1e-4f)
{
    m_sample_classes = new int[m_num_samples]{0};
    m_clusters = new DataType[m_num_clusters * m_num_features];
    for (int i = 0; i < this->m_num_clusters * this->m_num_features; ++i)
    {
        m_clusters[i] = h_clusters[i];
    }
    int data_buf_size = m_num_samples * m_num_features;
    int cluster_buf_size = m_num_clusters * m_num_features;
    int cache_size = m_num_samples;
    int mem_size = sizeof(DataType) * (data_buf_size + cluster_buf_size) + sizeof(int) * (m_num_samples) +
                   sizeof(float) * (m_num_samples + m_num_samples) + sizeof(int) * m_num_clusters + sizeof(float) * cache_size;

    CHECK_CUDA_ERROR(hipMalloc((void **)&d_data, mem_size));

    d_clusters = (DataType *)(d_data + data_buf_size);
    d_sample_classes = (int *)(d_clusters + cluster_buf_size);
    d_min_dist = (float *)(d_sample_classes + m_num_samples);
    d_loss = (float *)(d_min_dist + m_num_samples);
    d_cluster_size = (int *)(d_loss + m_num_samples);

    CHECK_CUDA_ERROR(hipMemcpy(d_clusters, h_clusters, sizeof(DataType) * cluster_buf_size, hipMemcpyHostToDevice));

    printf("num_samples: %d  num_clusters: %d  num_features: %d\n", num_samples, num_clusters, num_features);
}

template <typename DataType>
void KmeansGPU<DataType>::fit(const DataType *v_data)
{
    hipEvent_t start, stop;
    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&stop));
    CHECK_CUDA_ERROR(hipEventRecord(start));
    hipEventQuery(start);

    printf("***********************in fit*********************\n");
    float *h_loss = new float[m_num_samples]{0.0};

    hipStream_t calculate_stream, update_stream;
    CHECK_CUDA_ERROR(hipStreamCreate(&calculate_stream));
    CHECK_CUDA_ERROR(hipStreamCreate(&update_stream));

    hipEvent_t calculate_event, update_event;
    CHECK_CUDA_ERROR(hipEventCreate(&calculate_event));
    CHECK_CUDA_ERROR(hipEventCreate(&update_event));

    CHECK_CUDA_ERROR(hipMemcpyAsync(d_data, v_data, sizeof(DataType) * m_num_samples * m_num_features, hipMemcpyHostToDevice, calculate_stream));
    // CHECK_CUDA_ERROR(hipEventRecord(update_event, update_stream));
    char *d_cache_buf = (char *)(d_cluster_size + m_num_clusters);

    float lastLoss = 0.0f;
    for (int i = 0; i < m_max_iters; ++i)
    {
        launchFit<DataType>(d_data, d_clusters, d_sample_classes, d_cluster_size, d_min_dist, d_loss, d_cache_buf,
                            m_num_clusters, m_num_samples, m_num_features, calculate_stream, update_stream, 
                            calculate_event, update_event);

        CHECK_CUDA_ERROR(hipMemcpyAsync(h_loss, d_loss, sizeof(float) * m_num_samples, hipMemcpyDeviceToHost, calculate_stream));
        CHECK_CUDA_ERROR(hipStreamSynchronize(calculate_stream));
        this->m_optTarget = h_loss[0];
        if (std::abs(lastLoss - this->m_optTarget) < this->m_eplison) {
            printf("break!!!    lastLoss: %g  m_optTarget: %g\n", lastLoss, this->m_optTarget);
            break;
        }
        lastLoss = this->m_optTarget;
        printf("Iters: %d  current loss: %g\n", i+1, this->m_optTarget);
    }

    CHECK_CUDA_ERROR(hipEventRecord(stop));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    float elapsedTime;
    CHECK_CUDA_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
    CHECK_CUDA_ERROR(hipEventDestroy(start));
    CHECK_CUDA_ERROR(hipEventDestroy(stop));
    
    printf("Time = %g ms.\n", elapsedTime);


    CHECK_CUDA_ERROR(hipMemcpyAsync(m_clusters, d_clusters, sizeof(DataType) * m_num_clusters * m_num_features, hipMemcpyDeviceToHost, calculate_stream));
    CHECK_CUDA_ERROR(hipMemcpyAsync(m_sample_classes, d_sample_classes, sizeof(int) * m_num_samples, hipMemcpyDeviceToHost, calculate_stream));

    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    CHECK_CUDA_ERROR(hipEventDestroy(update_event));
    CHECK_CUDA_ERROR(hipEventDestroy(calculate_event));
    CHECK_CUDA_ERROR(hipStreamDestroy(calculate_stream));
    CHECK_CUDA_ERROR(hipStreamDestroy(update_stream));
    delete [] h_loss;
}

template <typename DataType>
KmeansGPU<DataType>::~KmeansGPU() 
{
    delete [] m_sample_classes;
    delete [] m_clusters;
    CHECK_CUDA_ERROR(hipFree(d_data));
}

template <typename DataType>
float KmeansGPU<DataType>::accuracy(int *label)
{
    float errCnt = 0;
    for (int i = 0; i < m_num_samples; ++i)
        if (m_sample_classes[i] != label[i])
            errCnt += 1.0;
    return 1.0 - errCnt / m_num_samples;
}


template class KmeansGPU<float>;

template class KmeansGPU<double>;

/*
template KmeansGPU<float>::KmeansGPU(float *h_clusters, int num_clusters, int num_features, int num_samples, int max_iters);
template void KmeansGPU<float>::fit(const float *v_data);

template KmeansGPU<double>::KmeansGPU(double *h_clusters, int num_clusters, int num_features, int num_samples, int max_iters);
template void KmeansGPU<double>::fit(const double *v_data);
*/

}   // clustering
